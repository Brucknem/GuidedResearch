#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

// // Kernel function to add the elements of two arrays
// __global__
// void add(int n, float *x, float *y)
// {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride)
//     y[i] = x[i] + y[i];
// }

// int main(void)
// {
//     int N = 1 << 20;
//     float *x, *y;

//     // Allocate Unified Memory – accessible from CPU or GPU
//     hipMallocManaged(&x, N * sizeof(float));
//     hipMallocManaged(&y, N * sizeof(float));

//     // initialize x and y arrays on the host
//     for (int i = 0; i < N; i++)
//     {
//         x[i] = 1.0f;
//         y[i] = 2.0f;
//     }

//     int blockSize = 256;
//     int numBlocks = (N + blockSize - 1) / blockSize;
//     add<<<numBlocks, blockSize>>>(N, x, y);

//     // Wait for GPU to finish before accessing on host
//     hipDeviceSynchronize();

//     // Check for errors (all values should be 3.0f)
//     float maxError = 0.0f;
//     for (int i = 0; i < N; i++){
//         maxError = fmax(maxError, fabs(y[i] - 3.0f));
//     }

//     std::cout << "Max error: " << maxError << std::endl;

//     // Free memory
//     hipFree(x);
//     hipFree(y);

//     return 0;
// }

#include "ceres/ceres.h"
#include "glog/logging.h"
using ceres::AutoDiffCostFunction;
using ceres::CostFunction;
using ceres::Problem;
using ceres::Solve;
using ceres::Solver;
// A templated cost functor that implements the residual r = 10 -
// x. The method operator() is templated so that we can then use an
// automatic differentiation wrapper around it to generate its
// derivatives.
struct CostFunctor {
  template <typename T>
  bool operator()(const T* const x, T* residual) const {
    residual[0] = 10.0 - x[0];
    return true;
  }
};
int main(int argc, char** argv) {
  google::InitGoogleLogging(argv[0]);
  // The variable to solve for with its initial value. It will be
  // mutated in place by the solver.
  double x = 0.5;
  const double initial_x = x;
  // Build the problem.
  Problem problem;
  // Set up the only cost function (also known as residual). This uses
  // auto-differentiation to obtain the derivative (jacobian).
  CostFunction* cost_function =
      new AutoDiffCostFunction<CostFunctor, 1, 1>(new CostFunctor);
  problem.AddResidualBlock(cost_function, nullptr, &x);
  // Run the solver!
  Solver::Options options;
  options.minimizer_progress_to_stdout = true;
  Solver::Summary summary;
  Solve(options, &problem, &summary);
  std::cout << summary.BriefReport() << "\n";
  std::cout << "x : " << initial_x << " -> " << x << "\n";
  return 0;
}